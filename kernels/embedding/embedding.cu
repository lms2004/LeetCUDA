#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

__global__ void embedding_f32_kernel(const int *idx, float *weight,
                                     float *output, int n, int emb_size) {
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
}

__global__ void embedding_f32x4_kernel(const int *idx, float *weight,
                                       float *output, int n, int emb_size) {
  int tx = threadIdx.x * 4;
  int bx = blockIdx.x;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
  output[bx * emb_size + tx + 1] = weight[offset + tx + 1];
  output[bx * emb_size + tx + 2] = weight[offset + tx + 2];
  output[bx * emb_size + tx + 3] = weight[offset + tx + 3];
}

__global__ void embedding_f32x4_pack_kernel(const int *idx, float *weight,
                                            float *output, int n,
                                            int emb_size) {
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  LDST128BITS(output[bx * emb_size + 4 * tx]) =
      LDST128BITS(weight[offset + 4 * tx]);
}

__global__ void embedding_f16_kernel(const int *idx, half *weight, half *output,
                                     int n, int emb_size) {
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
}

__global__ void emb_kernel_cu_fp32(int32_t vocab_size, int32_t token_num, int32_t weight_dim,
                                   const int32_t* input_ptr, const float* weight_ptr,
                                   float* output_ptr) {
  int32_t token_idx = blockIdx.x;
  if (token_idx >= token_num) {
    return;
  }
  int32_t token = input_ptr[token_idx];
  if (token >= vocab_size) {
    return;
  }

  float* output_ptr_start = output_ptr + token_idx * weight_dim;
  const float* weight_ptr_start = weight_ptr + token * weight_dim;

  for (int32_t i = threadIdx.x; i < weight_dim; i += blockDim.x) {
    output_ptr_start[i] = weight_ptr_start[i];
  }
}

__global__ void embedding_f16x8_kernel(const int *idx, half *weight,
                                       half *output, int n, int emb_size) {
  int tx = threadIdx.x * 8;
  int bx = blockIdx.x;
  int offset = idx[bx] * emb_size;
  output[bx * emb_size + tx] = weight[offset + tx];
  output[bx * emb_size + tx + 1] = weight[offset + tx + 1];
  output[bx * emb_size + tx + 2] = weight[offset + tx + 2];
  output[bx * emb_size + tx + 3] = weight[offset + tx + 3];
  output[bx * emb_size + tx + 4] = weight[offset + tx + 4];
  output[bx * emb_size + tx + 5] = weight[offset + tx + 5];
  output[bx * emb_size + tx + 6] = weight[offset + tx + 6];
  output[bx * emb_size + tx + 7] = weight[offset + tx + 7];
}

__global__ void embedding_f16x8_pack_kernel(const int *idx, half *weight,
                                            half *output, int n, int emb_size) {
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int tid = bx * blockDim.x + tx;
  int offset = idx[bx] * emb_size;
  LDST128BITS(output[bx * emb_size + 8 * tx]) =
      LDST128BITS(weight[offset + 8 * tx]);
}

// --------------------- PyTorch bindings for custom kernel
// -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                                    \
  if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) {                        \
    throw std::runtime_error("Tensor size mismatch!");                         \
  }

#define TORCH_BINDING_EMBEDDING(packed_type, th_type, element_type,            \
                                n_elements)                                    \
  void embedding_##packed_type(torch::Tensor a, torch::Tensor weight,          \
                               torch::Tensor o) {                              \
    CHECK_TORCH_TENSOR_DTYPE(a, (torch::kInt32));                              \
    CHECK_TORCH_TENSOR_DTYPE(weight, (th_type));                               \
    CHECK_TORCH_TENSOR_DTYPE(o, (th_type));                                    \
                                                                               \
    const int N = a.size(0);                                                   \
    const int emb_size = weight.size(1);                                       \
    dim3 block(emb_size / n_elements);                                         \
    dim3 grid(N);                                                              \
    embedding_##packed_type##_kernel<<<grid, block>>>(                         \
        reinterpret_cast<int *>(a.data_ptr()),                                 \
        reinterpret_cast<element_type *>(weight.data_ptr()),                   \
        reinterpret_cast<element_type *>(o.data_ptr()), N, emb_size);          \
  }

TORCH_BINDING_EMBEDDING(f32, torch::kFloat32, float, 1)
TORCH_BINDING_EMBEDDING(f32x4, torch::kFloat32, float, 4)
TORCH_BINDING_EMBEDDING(f32x4_pack, torch::kFloat32, float, 4)
TORCH_BINDING_EMBEDDING(f16, torch::kHalf, half, 1)
TORCH_BINDING_EMBEDDING(f16x8, torch::kHalf, half, 8)
TORCH_BINDING_EMBEDDING(f16x8_pack, torch::kHalf, half, 8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(embedding_f32);
  TORCH_BINDING_COMMON_EXTENSION(embedding_f32x4);
  TORCH_BINDING_COMMON_EXTENSION(embedding_f32x4_pack);
  TORCH_BINDING_COMMON_EXTENSION(embedding_f16);
  TORCH_BINDING_COMMON_EXTENSION(embedding_f16x8);
  TORCH_BINDING_COMMON_EXTENSION(embedding_f16x8_pack);
}
